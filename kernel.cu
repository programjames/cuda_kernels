#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace
{
    __global__ void _test_kernel(float *a, float *b, float *c, int n)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < n / 2)
        {
            // Multiply for the first half
            c[index] = a[index] * b[index];
        }
        else if (index < n)
        {
            // Add for the second half
            c[index] = a[index] + b[index];
        }
    }
}

size_t div_round_up(size_t x, size_t y)
{
    return (x + y - 1) / y;
}

void test_wrapper(const at::Tensor in_a,
                  const at::Tensor in_b,
                  at::Tensor out_c,
                  int block_size = 64)
{
    size_t N = in_a.numel();
    size_t num_blocks = div_round_up(N, block_size);
    _test_kernel<<<num_blocks, block_size>>>(
        in_a.data_ptr<float>(), in_b.data_ptr<float>(), out_c.data_ptr<float>(), N);
}