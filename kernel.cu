
#include <hip/hip_runtime.h>
extern "C" __global__ void multiply_add_kernel(float* a, float* b, float* c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n / 2) {
        // Multiply for the first half
        c[index] = a[index] * b[index];
    } else if (index < n) {
        // Add for the second half
        c[index] = a[index] + b[index];
    }
}